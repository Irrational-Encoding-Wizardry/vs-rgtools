#include "hip/hip_runtime.h"
__device__ static const int width = ${width};
__device__ static const int height = ${height};
__device__ static const float sigmaS = ${sigmaS};
__device__ static const float noise0 = ${noise0};
__device__ static const float noise1 = ${noise1};
__device__ static const int radius = ${radius};

__device__ static const int kernel_size_x = 2 * radius + ${block_x};
__device__ static const int kernel_size_y = 2 * radius + ${block_y};

extern "C"
__global__ void nabl(const ${data_type} * __restrict__ src, ${data_type} * __restrict__ dst) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;

    float num {};
    float den {};

    if (x >= width || y >= height)
        return;

    float center {};
    float value {};

    center = src[y * width + x];

    #pragma unroll 4
    for (int cy = max(y - radius, 0); cy <= min(y + radius, height - 1); ++cy) {
        #pragma unroll 4
        for (int cx = max(x - radius, 0); cx <= min(x + radius, width - 1); ++cx) {
            value = src[cy * width + cx];

            float noise_g = noise0 * value + noise1;
            float euc_dist = sqrtf((cy - y) * (cy - y) + (cx - x) * (cx - x));

            float weight = expf((euc_dist * euc_dist) / (-2 * (sigmaS * sigmaS))) * (
                expf(-2 * noise_g) * jnf(sqrtf((value - center) * value - center), 2 * noise_g)
            );

            num += weight * value;
            den += weight;
        }
    }

    dst[y * width + x] = num / den;
}